
#include "CUDA.cuh"

#include <iostream>
#include <sstream>
using namespace std;

// ----------------------------------------------------------------------------
// Diplay info about the specified device.
// ----------------------------------------------------------------------------
string open3d::DeviceInfo(const int& devID) {
    if (-1 == devID)
        return string();

    hipDeviceProp_t deviceProp;

    hipGetDeviceProperties(&deviceProp, devID);

    stringstream info;
    info << "GPU Device " << devID << ": ";
    info << deviceProp.name << ", ";
    info << "CUDA ";
    info << deviceProp.major << ".";
    info << deviceProp.minor << endl;

    return info.str();
}

// ---------------------------------------------------------------------------
// Alocate device memory and perform validation.
// ---------------------------------------------------------------------------
hipError_t open3d::AlocateDevMemory(double** d, const size_t& numElements,
    const int& devID) {
    hipError_t status = hipSuccess;

    size_t size = numElements * sizeof(double);

    status = hipMalloc((void **)d, size);

    return status;
}

// ---------------------------------------------------------------------------
// Copy data to the device.
// ---------------------------------------------------------------------------
hipError_t open3d::CopyHst2DevMemory(double* h, double* d,
    const size_t& numElements) {
    hipError_t status = hipSuccess;

    size_t size = numElements * sizeof(double);

    status = hipMemcpy(d, h, size, hipMemcpyHostToDevice);

    return status;
}

// ---------------------------------------------------------------------------
// Copy data from the device.
// ---------------------------------------------------------------------------
hipError_t open3d::CopyDev2HstMemory(double* d, double* h,
    const size_t& numElements) {
    hipError_t status = hipSuccess;

    size_t size = numElements * sizeof(double);

    status = hipMemcpy(h, d, size, hipMemcpyDeviceToHost);

    return status;
}

// ---------------------------------------------------------------------------
// Safely deallocate device memory.
// ---------------------------------------------------------------------------
hipError_t open3d::freeDev(double** d) {
    hipError_t status = hipSuccess;

    status = hipFree(*d);

    if (hipSuccess == status)
        *d = NULL;

    return status;
}

// ---------------------------------------------------------------------------
// update the device memory on demand
// ---------------------------------------------------------------------------
hipError_t open3d::UpdateDeviceMemory(double **d_data,
    const double* const data,
    const size_t& numElements,
    const int& devID) {
    hipError_t status = hipSuccess;

    if (*d_data != NULL) {
        status = hipFree(*d_data);
        if (hipSuccess != status) return status;

        *d_data = NULL;
    }

    size_t size = numElements * sizeof(double);

    hipSetDevice(devID);
    status = hipMalloc((void **)d_data, size);
    if (hipSuccess != status) return status;

    status = hipMemcpy(*d_data, data, size, hipMemcpyHostToDevice);
    if (hipSuccess != status) return status;

    return status;
}  // namespace geometry
