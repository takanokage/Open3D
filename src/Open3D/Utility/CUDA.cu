
#include "CUDA.cuh"

#include <iostream>
#include <sstream>
using namespace std;

// ----------------------------------------------------------------------------
// Diplay info about the specified device.
// ----------------------------------------------------------------------------
string open3d::DeviceInfo(const int& devID) {
    if (-1 == devID)
        return string();

    hipDeviceProp_t deviceProp;

    hipGetDeviceProperties(&deviceProp, devID);

    stringstream info;
    info << "GPU Device " << devID << ": ";
    info << deviceProp.name << ", ";
    info << "CUDA ";
    info << deviceProp.major << ".";
    info << deviceProp.minor << endl;

    return info.str();
}

// ----------------------------------------------------------------------------
// Display debug info.
// Requires building the project in debug mode.
// ----------------------------------------------------------------------------
void open3d::DebugInfo(const string& function_name, const hipError_t& status) {
    #ifndef NDEBUG
    if (hipSuccess != status) {
        string error_message = hipGetErrorString(status);
        printf("%20s: %s\n", function_name.c_str(), error_message.c_str());
    }
    #endif
}

// ---------------------------------------------------------------------------
// Alocate device memory and perform validation.
// ---------------------------------------------------------------------------
hipError_t open3d::AlocateDevMemory(double** d, const size_t& numElements,
    const int& devID) {
    hipError_t status = hipSuccess;

    if (CPU == devID) return status;

    size_t size = numElements * sizeof(double);

    hipSetDevice(devID);
    status = hipMalloc((void **)d, size);
    DebugInfo("AlocateDevMemory", status);

    return status;
}

// ---------------------------------------------------------------------------
// Copy data to the device.
// ---------------------------------------------------------------------------
hipError_t open3d::CopyHst2DevMemory(double* h, double* d,
    const size_t& numElements) {
    hipError_t status = hipSuccess;

    size_t size = numElements * sizeof(double);

    status = hipMemcpy(d, h, size, hipMemcpyHostToDevice);

    DebugInfo("CopyHst2DevMemory", status);

    return status;
}

// ---------------------------------------------------------------------------
// Copy data from the device.
// ---------------------------------------------------------------------------
hipError_t open3d::CopyDev2HstMemory(double* d, double* h,
    const size_t& numElements) {
    hipError_t status = hipSuccess;

    size_t size = numElements * sizeof(double);

    status = hipMemcpy(h, d, size, hipMemcpyDeviceToHost);

    DebugInfo("CopyDev2HstMemory", status);

    return status;
}

// ---------------------------------------------------------------------------
// Safely deallocate device memory.
// ---------------------------------------------------------------------------
hipError_t open3d::freeDev(double** d) {
    hipError_t status = hipSuccess;

    status = hipFree(*d);
    DebugInfo("freeDev", status);

    if (hipSuccess == status)
        *d = NULL;

    return status;
}

// ---------------------------------------------------------------------------
// update the device memory on demand
// ---------------------------------------------------------------------------
hipError_t open3d::UpdateDeviceMemory(double **d_data,
    const double* const data,
    const size_t& numElements,
    const int& devID) {
    hipError_t status = hipSuccess;

    if (CPU == devID) return status;

    if (*d_data != NULL) {
        status = hipFree(*d_data);
        DebugInfo("UpdateDeviceMemory", status);
        if (hipSuccess != status) return status;

        *d_data = NULL;
    }

    size_t size = numElements * sizeof(double);

    hipSetDevice(devID);
    status = hipMalloc((void **)d_data, size);
    DebugInfo("UpdateDeviceMemory", status);
    if (hipSuccess != status) return status;

    status = hipMemcpy(*d_data, data, size, hipMemcpyHostToDevice);
    DebugInfo("UpdateDeviceMemory", status);
    if (hipSuccess != status) return status;

    return status;
}  // namespace geometry
